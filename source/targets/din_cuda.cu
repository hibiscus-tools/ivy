#include "hip/hip_runtime.h"
#include <fmt/printf.h>

#define GLM_ENABLE_EXPERIMENTAL
#include <glm/glm.hpp>
#include <glm/gtc/random.hpp>
#include <glm/gtx/string_cast.hpp>

#include <littlevk/littlevk.hpp>

#include "core/texture.hpp"
#include "exec/globals.hpp"
#include "paths.hpp"

// TODO: specialization of typed texture is uint32_t for saving and loading...
template <typename T>
struct Texture {
	size_t width;
	size_t height;
	std::vector <T> pixels;

	T *operator[](size_t i) {
		return &pixels[i * width];
	}

	const T *operator[](size_t i) const {
		return &pixels[i * width];
	}

	T sample(glm::vec2 uv) const {
		uv = glm::clamp(uv, 0.0f, 1.0f);
		float i = (height - 1) * uv.x;
		float j = (width - 1) * uv.y;

		size_t i0 = floor(i);
		size_t i1 = ceil(i);

		size_t j0 = floor(j);
		size_t j1 = ceil(j);

		i -= i0;
		j -= j0;

		const T &v00 = pixels[i0 * width + j0];
		const T &v01 = pixels[i0 * width + j1];
		const T &v10 = pixels[i1 * width + j0];
		const T &v11 = pixels[i1 * width + j1];

		return v00 * (1 - i) * (1 - j)
			+ v01 * (1 - i) * j
			+ v10 * i * (1 - j)
			+ v11 * i * j;
	}

	ivy::Texture as_texture() const;

	static Texture from(size_t w, size_t h) {
		return { w, h, std::vector <T> (w * h) };
	}

	static Texture from(const ivy::Texture &texture) {
		return {
			(size_t) texture.width,
			(size_t) texture.height,
			texture.as_rgb()
		};
	}
};

template <typename T, typename U, typename F>
Texture <T> transform(const Texture <U> &texture, const F &ftn)
{
	auto result = Texture <T> ::from(texture.width, texture.height);
	for (size_t i = 0; i < texture.width * texture.height; i++)
		result.pixels[i] = ftn(texture.pixels[i]);

	return result;
}

template <>
ivy::Texture Texture <glm::vec4> ::as_texture() const
{
	std::vector <uint8_t> uint_pixels(width * height * sizeof(uint32_t));

	for (int i = 0; i < width; i++) {
		for (int j = 0; j < height; j++) {
			uint32_t index = (i * height + j);

			const glm::vec4 &c = pixels[index];

			uint8_t *dst = &uint_pixels[index << 2];
			dst[0] = 255.0f * c.r;
			dst[1] = 255.0f * c.g;
			dst[2] = 255.0f * c.b;
			dst[3] = 255.0f * c.a;
		}
	}

	return { (int) width, (int) height, 4, uint_pixels };
}

template <>
ivy::Texture Texture <glm::vec3> ::as_texture() const
{
	std::vector <uint8_t> uint_pixels(width * height * sizeof(uint32_t));

	for (int i = 0; i < width; i++) {
		for (int j = 0; j < height; j++) {
			uint32_t index = (i * height + j);

			const glm::vec3 &c = pixels[index];

			uint8_t *dst = &uint_pixels[index << 2];
			dst[0] = 255.0f * c.r;
			dst[1] = 255.0f * c.g;
			dst[2] = 255.0f * c.b;
			dst[3] = 0xff;
		}
	}

	return { (int) width, (int) height, 4, uint_pixels };
}

template <>
ivy::Texture Texture <glm::vec2> ::as_texture() const
{
	std::vector <uint8_t> uint_pixels(width * height * sizeof(uint32_t));

	for (int i = 0; i < width; i++) {
		for (int j = 0; j < height; j++) {
			uint32_t index = (i * height + j);

			const glm::vec2 &c = pixels[index];

			uint8_t *dst = &uint_pixels[index << 2];
			dst[0] = 255.0f * c.r;
			dst[1] = 255.0f * c.g;
			dst[2] = 0;
			dst[3] = 0xff;
		}
	}

	return { (int) width, (int) height, 4, uint_pixels };
}

struct DIn {
	Texture <glm::vec2> uvs;
	Texture <glm::vec3> colors;

	Texture <glm::vec3> render(size_t w, size_t h) const {
		auto tex = Texture <glm::vec3> ::from(w, h);

		for (size_t i = 0; i < h; i++) {
			for (size_t j = 0; j < w; j++) {
				glm::vec2 uv = { float(i)/h, float(j)/w };

				uv = uvs.sample(uv);
				tex[i][j] = colors.sample(uv);
			}
		}

		return tex;
	}

	static DIn from(size_t iw, size_t ih, size_t cw, size_t ch) {
		auto uvs = Texture <glm::vec2> ::from(iw, ih);
		auto colors = Texture <glm::vec3> ::from(cw, ch);

		// Normal UV initialization
		for (size_t i = 0; i < ih; i++) {
			for (size_t j = 0; j < iw; j++) {
				uvs[i][j] = {
					float(i)/ih,
					float(j)/iw
				};
			}
		}

		// Random color initialization
		for (size_t i = 0; i < ch; i++) {
			for (size_t j = 0; j < cw; j++)
				colors[i][j] = glm::linearRand(glm::vec3(0.0f), glm::vec3(1.0f));
		}

		return DIn { uvs, colors };
	}
};

int main()
{

}
